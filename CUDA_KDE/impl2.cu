#include "hip/hip_runtime.h"
//
//	Version information:
//
//	Contains normalising functions for hammingMat and waveData
//
//	MP			Initial version
//	BW			Added makeAllHamming() method
//				Added printMatDouble() to arrayOperations.cu file
//				Changed hammingMat size
//				Changed argument order in hipMemcpyDeviceToHost
//				Added sampleNum argument to hamming() method
//				
//	Grid for hamming matrix calculation
//	------------------------------------------------------------------  
//	|	256	|	256	|	256	|	256	|	256	|	256	| .....	|	256	|		
//	------------------------------------------------------------------
//	<-------------------------- 16 blocks ---------------------------->
//
//	Hamming result vector
//	--------------------------------------------------------------------
//	|<-200->			|   				|					|		
//	|   0  |...|   15   |		|	|		|		|	|		|	
//	|<-----16*200----->	|					|					|	
//	--------------------------------------------------------------------
// <-------------------------- 16*200*256 ---------------------------->
//	index of an element = key*KEYBYTES*SAMPLES + keyByte*SAMPLES + sampleNumber

#include <stdlib.h>
#include <stdio.h>
#include <syslog.h>
#include <limits.h>
#include "data.cuh"
#include "arrayOperations.cuh"
#include "probOps.cuh"
#include "helpers.cuh"

#define PLAINFILE "/storage/buddhi/data/plain.txt"
//#define WAVEFILE "wave.txt"
#define WAVEFILE "/storage/buddhi/data/wave.txt"

//number of sample points in one encryption
/*#define SAMPLEPOINTS 100000
#define SAMPLES 200

//defined for 128 bit AES
#define KEYBYTES 16
#define KEYS 256*/

//calculates hamming weight of a 8 bit number
__device__ byte hammingweight(byte H){

	// Count the number of set bits
	byte dist=0;
	while(H){
		dist++; 
		H &= H - 1;
	}
	return dist;
}

//find hamming weight for the selection function
/*__device__ byte hamming(unsigned int *sample, unsigned int i,unsigned int n,unsigned int key){ //n is byteno  i is the sample
    //byte inter = (byte)sbox[sample[i*KEYBYTES+n] ^ key];
    byte inter = (byte)sbox[sample[i] ^ key];
    //byte inter = 0x00;
    byte dist = hammingweight(inter);	  
	return dist;
}*/

__device__ byte hamming(unsigned int *sample, unsigned int index, unsigned int key, unsigned int sampleNum){ //n is byteno  i is the sample
	byte inter = sbox[sample[sampleNum*KEYBYTES+index] ^ key]; 
    //byte inter = 0xAB;
    byte dist = hammingweight(inter);	  
	return dist;
}

int arrayMax(unsigned int * data, int length){
	
	int max = data[0];
	for(int i=0; i<length; i++){
		if(data[i]>max)
			max = data[i];
	}
	return max;
}


/******************************************************************/
/**                   Kernels          							 **/
/******************************************************************/

//-------------------------------------------------------------------------------------------------------
//	Calculate all bandwidths for all keys and keybytes for hamming data
//  Calculate maximum and minimum of each data set
//-------------------------------------------------------------------------------------------------------
__global__ void KDE_bandwidthHammingKernel(double * dataArray, double * bandwidthArray, double * maxMinArray,
												int targetLengthH, double * targetArray){          
	
	/* Block size = KEYS/16, KEYS/16,1
	 * Grid size = KEYBYTE, 1,1
	 * */
	
	
	int j;
	int max = INT_MIN;
	int min = INT_MAX; 
	int keyByte = threadIdx.y;
	int key = blockIdx.x*blockDim.x + threadIdx.x;
	int targetLength = targetLengthH;
	double sum_array=0;
	double sum_2_array=0;

	
	
	if ((keyByte < KEYBYTES) && (key < KEYS)){
		for(j=0; j<SAMPLES; j++){

			int index = key*KEYBYTES*SAMPLES + keyByte*SAMPLES + j;
			sum_array += dataArray[index];
			sum_2_array += dataArray[index] * dataArray[index];
		
			if(dataArray[index] > max){
				max = dataArray[index];
			}
			
			if(dataArray[index] < min){
				min = dataArray[index];
			}
		
		}
		
		double x  = sum_array/SAMPLES;
		double x2 = sum_2_array/SAMPLES;
		//printf("%lf %lf ", x, x2);
		double sigma = sqrt(x2 - (x*x));
		double b = sigma*(pow((3.0*SAMPLES/4.0),(-1.0/5.0)));
		bandwidthArray[key*KEYBYTES + keyByte] = b;
		
		maxMinArray[key*2*KEYBYTES+2*keyByte] = min;
		maxMinArray[key*2*KEYBYTES+2*keyByte+1] = max;
		
		/* **************** calculate target arrays ***************** */
		
		double x_max = max + (3*b); 
		double x_min = min - (3*b); 	
		double x_increment =  (x_max - x_min)/targetLength;		
		double xVal=x_min;
		j=0;
		for(xVal = x_min; xVal<x_max; j++,xVal += x_increment){
			if(j<targetLength){
				
				int index = key*KEYBYTES*targetLength + keyByte*targetLength+j;
				targetArray[index] = xVal;
				if((keyByte == 10) && (key == 255))
					printf("%d\n", j);
					
			}
			
		}
		
		
	}
}

//-------------------------------------------------------------------------------------------------------
//	Calculate all bandwidths for all samplepoints for wave data
//-------------------------------------------------------------------------------------------------------
__global__ void KDE_bandwidthWaveKernel(double * dataArray, double * bandwidthArray, double * maxMinArray, 
											int targetLengthW, double * targetArray){
	
	/* block Size = SAMPLEPOINTS/1000,1,1
	 * grid Size = SAMPLEPOINTS/100,1,1
	 * */
	
	
	int j;
	int max = INT_MIN;
	int min = INT_MAX; 
	int samplePoint = blockIdx.x*blockDim.x + threadIdx.x;
	int targetLength = targetLengthW;
	double sum_array=0;
	double sum_2_array=0;

	if (samplePoint < SAMPLEPOINTS){
		
	
		
		for(j=0; j<SAMPLES; j++){
			int index = samplePoint*SAMPLES + j;
			sum_array += dataArray[index];
			sum_2_array += dataArray[index] * dataArray[index];
			
			//if(isnan(sum_2_array)!=0){ //x2 ERROR!!
			//printf("error at: %d\n", samplePoint);
				//__threadfence();
				//asm("trap;");
			//}
			
			if(dataArray[index] > max){
				max = dataArray[index];
			}
			
			if(dataArray[index] < min){
				min = dataArray[index];
			}
		
		}
		
		double x  = sum_array/SAMPLES;
		double x2 = sum_2_array/SAMPLES;
		
		double sigma = sqrt(x2 - (x*x));
		
		double b = sigma*(pow((3.0*SAMPLES/4.0),(-1.0/5.0)));
		bandwidthArray[samplePoint] = b;
		
		//if(isnan(x2)!=0){ //x2 ERROR!!
			//printf("error at: %d\n", samplePoint);
			//__threadfence();
			//asm("trap;");
		//}
		
		maxMinArray[2*samplePoint] = min;
		maxMinArray[2*samplePoint+1] = max;
		
		/* **************** calculate target arrays ***************** */
		
		double x_max = max + (3*b); 
		double x_min = min - (3*b); 	
		double x_increment =  (x_max - x_min)/targetLength;		
		double xVal=x_min;
		j=0;
		for(xVal = x_min; xVal<x_max; j++,xVal += x_increment){
			if(j<targetLength){
				int index = samplePoint*targetLength+j;
				targetArray[index] = xVal;
			}
		}
		
		
	}
}

//----------------------------------------------------------------------------------------------------------
//	Method to calculate hamming weights of all keyBytes of all samples w.r.t. all key guesses
//----------------------------------------------------------------------------------------------------------
__global__ void makeAllHamming(unsigned int *plaintxt, double *hammingRes, int n){//n is the size of samples 
	
	int keyByte = threadIdx.y;
	int key = blockIdx.x*KEYBYTES + threadIdx.x;
	int i;

	if ((keyByte < KEYBYTES) && (key < KEYS)){
		for (i=0; i<SAMPLES; i++){
			double temp = (double)hamming(plaintxt,keyByte, key, i); 			
			hammingRes[key*KEYBYTES*SAMPLES + keyByte*SAMPLES + i] = temp;
		}
	}
} 

//--------------------------------------------------------------------------------------------------------
//	Sort each MI set
//--------------------------------------------------------------------------------------------------------
__global__ void simpleSort(double *origMat, int cols, double *maxOfEach, int key){
	
	int index = threadIdx.x;
	int i;
	double max = 0.0;
	for (i=0;i<cols;i++){
		if (max<origMat[key*KEYBYTES*cols + index*cols + i])
			max = origMat[key*KEYBYTES*cols + index*cols + i];
	}
	maxOfEach[key*KEYBYTES + index] = max;
}

int main(){

	//hipSetDevice(1);
	//int nDevices;
	
	//hipGetDeviceCount(&nDevices);
	//for (int i = 0; i < nDevices; i++) {
    //hipDeviceProp_t prop;
    //hipGetDeviceProperties(&prop, i);
    //printf("Device Number: %d\n", i);
    //printf("  Device name: %s\n", prop.name);
    //printf("  Max Blocks: %d\n",
    //       *prop.maxGridSize);
  //}


	//plainText at host
	int width = 256*16;
	int i,j;
	unsigned int *dev_plainTexts;
	unsigned int *plainTexts = (unsigned int*)malloc(KEYBYTES*SAMPLES*sizeof(unsigned int));
	//checkMalloc(plainTexts);

	/*** Reading plaintext file ***/
	FILE *plainT = fopen(PLAINFILE,"r");
	char temp1[32];	

	for(i=0; i<SAMPLES ;i++){
		fscanf(plainT,"%s",temp1);
		for(j=0; j<KEYBYTES; j++){
			int post = i*KEYBYTES + j;
			sscanf(&temp1[2*j],"%02X",&plainTexts[post]);		
		}
	}
	
	/*** reading wave file ***/
	FILE *waveStuff = fopen(WAVEFILE,"r");
	
	double *waveData= (double *)malloc(sizeof(double) * SAMPLEPOINTS * SAMPLES);

	float dat;

	
	for(i=0; i<SAMPLES ;i++){
		for(j=0; j<SAMPLEPOINTS; j++){
			fscanf(waveStuff,"%f",&dat); //?? 
			int pos = j*SAMPLES + i;
			//waveData[j][i]=(double)dat*1000.0;
			waveData[pos]=(double)dat*1000.0;
		}
	}
	
	fclose(waveStuff);
	
	//openlog("CUDAmia", NULL, 0);
	printf("WIDTH: %d\n", WIDTH);
	/****************************Calling makeAllHamming kernel*************************/
	
	//hammingMat at host
	//double *hammingMat = (double *)malloc(KEYBYTES*SAMPLES*sizeof(double));
	double *hammingMat = (double *)malloc(KEYS*KEYBYTES*SAMPLES*sizeof(double));
	double *dev_hammingMat;
	hipMalloc((void **)&dev_plainTexts, KEYBYTES*SAMPLES*sizeof(unsigned int));
	hipMemcpy(dev_plainTexts, plainTexts, KEYBYTES*SAMPLES*sizeof(unsigned int), hipMemcpyHostToDevice);
	//hipMalloc((void **)&dev_hammingMat, KEYBYTES*SAMPLES*sizeof(double));
	hipMalloc((void **)&dev_hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double));
	
	dim3 numBlocks(16,1); //Blocks in the grid
	dim3 numThreads(16,16); // Threads per block
	
	makeAllHamming<<<numBlocks,numThreads>>>(dev_plainTexts, dev_hammingMat, width);
	hipDeviceSynchronize();
	
	//hipMemcpy(hammingMat, dev_hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double), hipMemcpyDeviceToHost);
	
	//hipFree(dev_hammingMat);
	hipFree(dev_plainTexts);
	printf("make hamming done\n");
	
	/*****************************************************************************************
	 ******************************************************************************************/
	

	 /* **************************** KDE_bandwidthHammingKernel ***************************** */
	int targetLengthH = 12; 
	double *dev_banwWidthHamming;
	hipMalloc((void **)&dev_banwWidthHamming, KEYBYTES*KEYS*sizeof(double)); checkCudaError();
	
	double *dev_targetHamming;
	hipMalloc((void **)&dev_targetHamming, KEYBYTES*KEYS*targetLengthH*sizeof(double)); checkCudaError();
	
	double *dev_maxMinHamming;
	hipMalloc((void **)&dev_maxMinHamming, KEYBYTES*KEYS*2*sizeof(double)); checkCudaError();
	
	
	
	double *banwWidthHamming = (double *)malloc(KEYS*KEYBYTES*sizeof(double));
	double *targetHamming = (double *)malloc(KEYBYTES*KEYS*targetLengthH*sizeof(double));
	
	dim3 numBlocksBWH(16,1); //Blocks in the grid
	dim3 numThreadsBWH(16,16); // Threads per block
	
	KDE_bandwidthHammingKernel<<<numBlocksBWH, numThreadsBWH>>>(dev_hammingMat, dev_banwWidthHamming, dev_maxMinHamming,
																targetLengthH, dev_targetHamming); checkCudaError();
	//SEEMS FINE!!															
	hipMemcpy(banwWidthHamming, dev_banwWidthHamming, KEYBYTES*KEYS*sizeof(double), hipMemcpyDeviceToHost);checkCudaError();
	
	/*FILE * fpP = freopen("results.txt", "w", stdout);
	int ii, jj;
	for(ii=0;ii<KEYS;ii++){
		for(jj=0;jj<KEYBYTES;jj++){
			int post = ii*KEYBYTES+jj;
			printf("%lf  ",banwWidthHamming[post]);
		}
			printf("\n");
	}
	fclose(fpP);*/
	
	printf("bandwidth Hamming done\n");
	free(banwWidthHamming);
	free(targetHamming);
	/* **************************** KDE_bandwidthWaveKernel ***************************** */
	int targetLengthW = 256; 
	double * dev_waveData;
	hipMalloc((void **)&dev_waveData, SAMPLEPOINTS * SAMPLES*sizeof(double)); checkCudaError();
	hipMemcpy(dev_waveData, waveData, SAMPLEPOINTS * SAMPLES*sizeof(double), hipMemcpyHostToDevice);checkCudaError();
	
	double *dev_banwWidthWave;
	hipMalloc((void **)&dev_banwWidthWave, SAMPLEPOINTS*sizeof(double)); checkCudaError();
	
	double *dev_targetWave;
	hipMalloc((void **)&dev_targetWave, SAMPLEPOINTS*targetLengthW*sizeof(double)); checkCudaError();
	
	double *dev_maxMinWave;
	hipMalloc((void **)&dev_maxMinWave, SAMPLEPOINTS*2*sizeof(double)); checkCudaError();

	double *banwWidthWave = (double *)malloc(SAMPLEPOINTS*sizeof(double));
	double *targetWave = (double *)malloc(SAMPLEPOINTS*targetLengthW*sizeof(double));
	
	dim3 numBlocksBWW(100, 1, 1);
	dim3 numThreadsBWW(1000, 1, 1);
	
	KDE_bandwidthWaveKernel<<<numBlocksBWW, numThreadsBWW>>>(dev_waveData, dev_banwWidthWave, dev_maxMinWave,
																targetLengthW, dev_targetWave);checkCudaError();
	hipMemcpy(banwWidthWave, dev_banwWidthWave, SAMPLEPOINTS*sizeof(double), hipMemcpyDeviceToHost);checkCudaError();
	
	/*FILE * fpP = freopen("results.txt", "w", stdout);
	int ii, jj;
	for(ii=0;ii<SAMPLEPOINTS;ii++){
		//for(jj=0;jj<SAMPLES;jj++){
			//int post = ii*SAMPLES+jj;
			printf("%lf  ",banwWidthWave[ii]);
		//}
			printf("\n");
	}
	fclose(fpP);*/
	printf("bandwidth Wave done\n");
	free(banwWidthWave);
	/* **************************** KDE_findProbsHamming ******************************** */
	
	double *dev_hammingDataProbs;
	hipMalloc((void **)&dev_hammingDataProbs, KEYBYTES*KEYS*targetLengthH*sizeof(double)); checkCudaError();
	
	double *hammingDataProbs = (double *)malloc(KEYBYTES*KEYS*targetLengthH*sizeof(double));
	
	dim3 numBlocksProbsHKDE(256, 1, 1); 
	dim3 threadsPerBlocksProbsHKDE(16,1,1);
	
	KDE_findProbsHamming<<<numBlocksProbsHKDE, threadsPerBlocksProbsHKDE>>>(dev_hammingMat, dev_hammingDataProbs, 
							dev_banwWidthHamming, targetLengthH, dev_maxMinHamming);
	
	//hipMemcpy(hammingDataProbs, dev_hammingDataProbs, KEYBYTES*KEYS*targetLengthH*sizeof(double), hipMemcpyDeviceToHost);
	
	//FILE * fpP = freopen("results.txt", "w", stdout);
	//printMatDouble(hammingDataProbs, KEYS, KEYBYTES*targetLengthH);
	printf("prob hamming done\n");
	//fclose(fpP);
	free(hammingDataProbs);
	
	/* **************************** KDE_findProbsWave ******************************** */
	
	double *dev_waveDataProbs;
	hipMalloc((void **)&dev_waveDataProbs, SAMPLEPOINTS*targetLengthW*sizeof(double)); checkCudaError();
	
	double *waveDataProbs = (double *)malloc(SAMPLEPOINTS*targetLengthW*sizeof(double));
	
	dim3 numBlocksProbsWKDE(16, 16, 1); 
	dim3 threadsPerBlocksProbsWKDE(512, 1, 1);
	
	KDE_findProbsWave<<<numBlocksProbsWKDE, threadsPerBlocksProbsWKDE>>>(dev_waveData, dev_waveDataProbs, 
							dev_banwWidthWave, targetLengthW, dev_maxMinWave);
	checkCudaError();						
	//hipMemcpy(waveDataProbs, dev_waveDataProbs, SAMPLEPOINTS*targetLengthW*sizeof(double), hipMemcpyDeviceToHost);
	
	//FILE * fpP = freopen("results.txt", "w", stdout);
	//printMatDouble(waveDataProbs, SAMPLEPOINTS, targetLengthW);
	printf("prob wave done\n");
	//fclose(fpP);
	free(waveDataProbs);
	
	hipFree(dev_maxMinWave);
	hipFree(dev_maxMinHamming);
	
	/* **************************** KDE_findProbsJoint2 ******************************** */
	int targetLengthJoint = targetLengthW*targetLengthH;
	double *dev_jointProbs;
	hipMalloc((void **)&dev_jointProbs, (SAMPLEPOINTS/DIVIDE)*targetLengthJoint*sizeof(double)); checkCudaError();
	
	double *jointProbs = (double *)malloc((SAMPLEPOINTS/DIVIDE)*targetLengthJoint*sizeof(double));checkCudaError();
	
	int turn = 0;
	int key = 0;
	int keyByte = 0;
	
	dim3 blocksProbsJointKDE(ceil(targetLengthH/4.0),ceil(targetLengthW/16.0),SAMPLEPOINTS/(DIVIDE*4));
	//dim3 blocksProbsJointKDE(3,16,12500);
	dim3 threadsProbsJointKDE(4,16,4);
	
	/* KDE_MI kernel */
	dim3 blocksMIJointKDE(200,0,0);
	dim3 threadsProbsJointKDE(256,0,0);
	
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
	
	for(key=0; key<1; key++){
		for(i=0; i<1; i++){
			
			KDE_findJointProbs2<<<blocksProbsJointKDE, threadsProbsJointKDE>>>(dev_hammingMat, dev_waveData, targetLengthW, 
									targetLengthH, dev_banwWidthWave, dev_banwWidthHamming, dev_targetHamming, dev_targetWave, 
									dev_jointProbs,turn, key, keyByte);
			hipDeviceSynchronize();						
			checkCudaError();
			
			turn = turn + 50000;
			//hipMemset(dev_jointProbs, 0, (SAMPLEPOINTS/DIVIDE)*targetLengthJoint*sizeof(double)); checkCudaError();
			
			hipMemcpy(jointProbs, dev_jointProbs, (SAMPLEPOINTS/DIVIDE)*targetLengthJoint*sizeof(double), hipMemcpyDeviceToHost);
			checkCudaError();
			
		}
	}
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(stdout,"Time spent for operation : %.10f seconds\n",elapsedtime/(float)1000);
	
	FILE * fpP = freopen("results.txt", "w", stdout);
	int ii, jj;
	for(ii=0;ii<SAMPLEPOINTS/DIVIDE;ii++){
		for(jj=0;jj<targetLengthJoint;jj++){
			int post = ii*targetLengthJoint+jj;
			printf("%lf  ",jointProbs[post]);
		}
			printf("\n");
	}
	fclose(fpP);
	free(jointProbs);
	printf("prob joint done\n");

	hipFree(dev_waveDataProbs);
	hipFree(dev_hammingDataProbs);
	hipFree(dev_banwWidthWave);
	hipFree(dev_banwWidthHamming);
	hipFree(dev_hammingMat);
	hipFree(dev_waveData);
	hipFree(dev_targetWave);
	hipFree(dev_targetHamming);
	hipFree(dev_jointProbs);
	
	/*** NO ERRORS!! ***/
	
	/*****************************************************************************************
	 ******************************************************************************************/	
	
	
	/****************** Calling normaliseWaveDataKernel on waveData ********************/
	/*
	//Store wave data in device
	//double * dev_waveData;
	//hipMalloc((void **)&dev_waveData, SAMPLEPOINTS * SAMPLES*sizeof(double));
	//hipMemcpy(dev_waveData, waveData, SAMPLEPOINTS * SAMPLES*sizeof(double), hipMemcpyHostToDevice);
	
	//Store normalised wave data
	unsigned int * waveDataNormalised = (unsigned int *)malloc(sizeof(unsigned int) * SAMPLEPOINTS * SAMPLES);
	unsigned int * dev_waveDataNormalised;
	hipMalloc((void **)&dev_waveDataNormalised, SAMPLEPOINTS * SAMPLES*sizeof(unsigned int));
	
	//Store firstNumState
	unsigned int * dev_firstNumState;
	hipMalloc((void **)&dev_firstNumState, sizeof(unsigned int)*SAMPLEPOINTS);
	unsigned int * firstNumState = (unsigned int *)malloc(sizeof(unsigned int) * SAMPLEPOINTS);
	
	dim3 numBlocksNorm(100, 1, 1);
	dim3 numThreadsNorm(1000, 1, 1);
	
	normaliseWaveDataKernel<<<numBlocksNorm, numThreadsNorm>>>(dev_waveData, dev_waveDataNormalised, dev_firstNumState);
	hipDeviceSynchronize();
	
	hipMemcpy(waveDataNormalised, dev_waveDataNormalised, SAMPLEPOINTS*SAMPLES*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(firstNumState, dev_firstNumState, SAMPLEPOINTS*sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	hipFree(dev_waveData);
	//hipFree(dev_waveDataNormalised);
	
	printf("normalise wave done\n");*/
	
	/***********************Calling normalisingHammingKernel*********************************/
	/*
	//Hamming data
	double * dev_hamming;
	hipMalloc((void **)&dev_hamming, KEYS*KEYBYTES*SAMPLES*sizeof(double));
	hipMemcpy(dev_hamming, hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double), hipMemcpyHostToDevice);
	
	//Result matrices
	unsigned int * hammingNormalised = (unsigned int *)malloc(sizeof(unsigned int) * KEYS*KEYBYTES*SAMPLES);
	unsigned int * dev_hammingNormalised;
	hipMalloc((void **)&dev_hammingNormalised, KEYS*KEYBYTES*SAMPLES*sizeof(unsigned int));

	//Store secondNumState
	unsigned int * dev_secondNumState;
	hipMalloc((void **)&dev_secondNumState, sizeof(unsigned int)*KEYS*KEYBYTES);
	unsigned int * secondNumState = (unsigned int *)malloc(sizeof(unsigned int) * KEYS*KEYBYTES);
	
	dim3 numBlocksHamming(256, 1, 1);
	dim3 numThreadsHamming(16, 1, 1);
	
	normaliseHammingKernel<<<numBlocksHamming, numThreadsHamming>>>(dev_hamming, dev_hammingNormalised, dev_secondNumState);
	hipDeviceSynchronize();
	
	hipMemcpy(hammingNormalised, dev_hammingNormalised, KEYS*KEYBYTES*SAMPLES*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(secondNumState, dev_secondNumState, KEYS*KEYBYTES*sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	hipFree(dev_hamming);
	printf("Hamming normalised, dev_hamming removed\n");*/
	
	/***********Sorting***************/
	/*
	double *dev_sortedMI;
	hipMalloc((void **)&dev_sortedMI, KEYBYTES*KEYS*sizeof(double)); 
	hipMemset(dev_sortedMI, 0, KEYBYTES*KEYS*sizeof(double));
	double *sortedMI = (double *)malloc(KEYBYTES*KEYS*sizeof(double));
	
	dim3 numBlockSort(1,1,1);
	dim3 threadsPerBlockSort(16,1,1);
	
	for (i=0;i<NUMOFKEYS;i++){
		simpleSort<<<numBlockSort, threadsPerBlockSort>>>(dev_MIvals, 100000, dev_sortedMI, i); checkCudaError();
	}
	hipMemcpy(sortedMI, dev_sortedMI, KEYBYTES*KEYS*sizeof(double), hipMemcpyDeviceToHost); checkCudaError();
	
	
	
	FILE * fpP = freopen("results_MISorted.txt", "w", stdout);
	int pp1,pp2;
	for(pp1=0;pp1<NUMOFKEYS;pp1++){
		for(pp2=0;pp2<KEYBYTES;pp2++){
			printf("%lf  ",sortedMI[pp1*KEYBYTES + pp2]);
		}
		printf("\n");
	}
	fclose(fpP);*/
	/****************************************************************************************************************************/
	//free(firstStateProbs);
	//free(secondStateProbs);
	//free(jointProbs);
	
	//firstStateProbs = NULL;
	//secondStateProbs = NULL;
	//jointProbs = NULL;
	
	free(plainTexts);
	free(hammingMat);
	free(waveData);
	//free(waveDataNormalised);
	//free(hammingNormalised);
	//free(MIvals);
	//free(sortedMI);
	
	/*hipFree(dev_firstStateProbs);
	hipFree(dev_secondStateProbs);
	
	hipFree(dev_jointProbs);
	
	hipFree(dev_waveDataNormalised);
	printf("dev_waveDataNormalised done\n");

	hipFree(dev_hammingNormalised);
	printf("dev_hammingNormalised done\n");

	hipFree(dev_firstNumState);
	printf("dev_firstNumState done\n");

	hipFree(dev_secondNumState);
	printf("dev_secondNumState done\n");
	
	hipFree(dev_MIvals);
	printf("dev_secondNumState done\n");
	printf("*********************END OF PROGRAM*********************\n");
	
	hipFree(dev_sortedMI);*/
	
	return 0;
}
