#include "hip/hip_runtime.h"
//
//	Version information:
//
//	Contains normalising functions for hammingMat and waveData
//
//	MP			Initial version
//	BW			Added makeAllHamming() method
//				Added printMatDouble() to arrayOperations.cu file
//				Changed hammingMat size
//				Changed argument order in hipMemcpyDeviceToHost
//				Added sampleNum argument to hamming() method
//				
//	Grid for hamming matrix calculation
//	------------------------------------------------------------------  
//	|	256	|	256	|	256	|	256	|	256	|	256	| .....	|	256	|		
//	------------------------------------------------------------------
//	<-------------------------- 16 blocks ---------------------------->
//
//	Hamming result vector
//	--------------------------------------------------------------------
//	|<-200->			|   				|					|		
//	|   0  |...|   15   |		|	|		|		|	|		|	
//	|<-----16*200----->	|					|					|	
//	--------------------------------------------------------------------
// <-------------------------- 16*200*256 ---------------------------->
//	index of an element = key*KEYBYTES*SAMPLES + keyByte*SAMPLES + sampleNumber

#include <stdlib.h>
#include <stdio.h>
#include <syslog.h>
#include "data.cuh"
#include "arrayOperations.cuh"
#include "probOps.cuh"
#include "helpers.cuh"

#define PLAINFILE "plain.txt"
//#define WAVEFILE "wave.txt"
#define WAVEFILE "/home/e11444/CO421/test2/wave.txt"

//number of sample points in one encryption
/*#define SAMPLEPOINTS 100000
#define SAMPLES 200

//defined for 128 bit AES
#define KEYBYTES 16
#define KEYS 256*/

//calculates hamming weight of a 8 bit number
__device__ byte hammingweight(byte H){

	// Count the number of set bits
	byte dist=0;
	while(H){
		dist++; 
		H &= H - 1;
	}
	return dist;
}

//find hamming weight for the selection function
/*__device__ byte hamming(unsigned int *sample, unsigned int i,unsigned int n,unsigned int key){ //n is byteno  i is the sample
    //byte inter = (byte)sbox[sample[i*KEYBYTES+n] ^ key];
    byte inter = (byte)sbox[sample[i] ^ key];
    //byte inter = 0x00;
    byte dist = hammingweight(inter);	  
	return dist;
}*/

__device__ byte hamming(unsigned int *sample, unsigned int index, unsigned int key, unsigned int sampleNum){ //n is byteno  i is the sample
	byte inter = sbox[sample[sampleNum*KEYBYTES+index] ^ key]; 
    //byte inter = 0xAB;
    byte dist = hammingweight(inter);	  
	return dist;
}

//__global__ void makeHamming(unsigned int *plaintxt, double *hammingRes, int n){//n is the size of samples 
	
	//unsigned int key = 0x01;
	//int index = blockIdx.x*KEYBYTES + threadIdx.x;
	
	//if (index<n){
		//byte temp = hamming(plaintxt,index, key);
		//hammingRes[index] = temp;
	//}
//} 

//----------------------------------------------------------------------------------------------------------
//	Method to calculate hamming weights of all keyBytes of all samples w.r.t. all key guesses
//----------------------------------------------------------------------------------------------------------
__global__ void makeAllHamming(unsigned int *plaintxt, double *hammingRes, int n){//n is the size of samples 
	
	int keyByte = threadIdx.y;
	int key = blockIdx.x*KEYBYTES + threadIdx.x;
	int i;

	if ((keyByte < KEYBYTES) && (key < KEYS)){
		
		for (i=0; i<SAMPLES; i++){
			double temp = (double)hamming(plaintxt,keyByte, key, i); 			
			hammingRes[key*KEYBYTES*SAMPLES + keyByte*SAMPLES + i] = temp;
		}
	}
} 

//--------------------------------------------------------------------------------------------------------
//	Find maximum of each MI set
//--------------------------------------------------------------------------------------------------------
__global__ void simpleSort(double *origMat, int cols, double *maxOfEach){
	
	int xx = blockIdx.x*blockDim.x + threadIdx.x;//16
	int yy = blockIdx.y*blockDim.y + threadIdx.y;//256
	
	int i;
	double max = 0.0;
	for (i=0;i<cols;i++){
		int pos = yy*KEYBYTES*SAMPLEPOINTS + xx*SAMPLEPOINTS + i;
		if (max<origMat[pos])
			max = origMat[pos];
	}
	maxOfEach[yy*KEYBYTES + xx] = max;
}

int arrayMax(unsigned int * data, int length){
	
	int max = data[0];
	for(int i=0; i<length; i++){
		if(data[i]>max)
			max = data[i];
	}
	return max;
}

int main(){

	//hipSetDevice(1);
	


	//plainText at host
	int width = 256*16;
	int i,j;
	
	unsigned int *plainTexts = (unsigned int*)malloc(KEYBYTES*SAMPLES*sizeof(unsigned int));
	//checkMalloc(plainTexts);

	/*** Reading plaintext file ***/
	FILE *plainT = fopen(PLAINFILE,"r");
	char temp1[32];	

	for(i=0; i<SAMPLES ;i++){
		fscanf(plainT,"%s",temp1);
		for(j=0; j<KEYBYTES; j++){
			int post = i*KEYBYTES + j;
			sscanf(&temp1[2*j],"%02X",&plainTexts[post]);		
		}
	}
	
	/*** reading wave file ***/
	FILE *waveStuff = fopen(WAVEFILE,"r");
	
	double *waveData= (double *)malloc(sizeof(double) * SAMPLEPOINTS * SAMPLES);

	float dat;

	
	for(i=0; i<SAMPLES ;i++){
		for(j=0; j<SAMPLEPOINTS; j++){
			fscanf(waveStuff,"%f",&dat); //?? 
			int pos = j*SAMPLES + i;
			//waveData[j][i]=(double)dat*1000.0;
			waveData[pos]=(double)dat*1000.0;
		}
	}
	
	fclose(waveStuff);
	
	//openlog("CUDAmia", NULL, 0);
	printf("WIDTH: %d\n", WIDTH);
	/****************************Calling makeAllHamming kernel*************************/
	
	//hammingMat at host
	//double *hammingMat = (double *)malloc(KEYBYTES*SAMPLES*sizeof(double));
	double *hammingMat = (double *)malloc(KEYS*KEYBYTES*SAMPLES*sizeof(double));
	
	
	unsigned int*dev_plainTexts;
	hipMalloc((void **)&dev_plainTexts, KEYBYTES*SAMPLES*sizeof(unsigned int));	checkCudaError();
	hipMemcpy(dev_plainTexts, plainTexts, KEYBYTES*SAMPLES*sizeof(unsigned int), hipMemcpyHostToDevice);	checkCudaError();
	
	double *dev_hammingMat;
	hipMalloc((void **)&dev_hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double));	checkCudaError();
	
	dim3 numBlocks(16,1); //Blocks in the grid
	dim3 numThreads(16,16); // Threads per block
	
	makeAllHamming<<<numBlocks,numThreads>>>(dev_plainTexts, dev_hammingMat, width);
	hipDeviceSynchronize();	checkCudaError();
	
	hipMemcpy(hammingMat, dev_hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double), hipMemcpyDeviceToHost);	checkCudaError();
	
	hipFree(dev_hammingMat);	checkCudaError();
	hipFree(dev_plainTexts);	checkCudaError();
	printf("make hamming done\n");
	
	/****************** Calling normaliseWaveDataKernel on waveData ********************/

	//Store wave data in device
	double * dev_waveData;
	hipMalloc((void **)&dev_waveData, SAMPLEPOINTS * SAMPLES*sizeof(double));	checkCudaError();
	hipMemcpy(dev_waveData, waveData, SAMPLEPOINTS * SAMPLES*sizeof(double), hipMemcpyHostToDevice);	checkCudaError();
	
	//Store normalised wave data
	unsigned int * waveDataNormalised = (unsigned int *)malloc(sizeof(unsigned int) * SAMPLEPOINTS * SAMPLES);
	unsigned int * dev_waveDataNormalised;
	hipMalloc((void **)&dev_waveDataNormalised, SAMPLEPOINTS * SAMPLES*sizeof(unsigned int));	checkCudaError();
	
	//Store firstNumState
	unsigned int * dev_firstNumState;
	hipMalloc((void **)&dev_firstNumState, sizeof(unsigned int)*SAMPLEPOINTS);	checkCudaError();
	
	unsigned int * firstNumState = (unsigned int *)malloc(sizeof(unsigned int) * SAMPLEPOINTS);
	
	dim3 numBlocksNorm(100, 1, 1);
	dim3 numThreadsNorm(1000, 1, 1);
	
	normaliseWaveDataKernel<<<numBlocksNorm, numThreadsNorm>>>(dev_waveData, dev_waveDataNormalised, dev_firstNumState);
	hipDeviceSynchronize();	checkCudaError();
	
	hipMemcpy(waveDataNormalised, dev_waveDataNormalised, SAMPLEPOINTS*SAMPLES*sizeof(unsigned int), hipMemcpyDeviceToHost);	checkCudaError();
	hipMemcpy(firstNumState, dev_firstNumState, SAMPLEPOINTS*sizeof(unsigned int), hipMemcpyDeviceToHost);	checkCudaError();
	
	hipFree(dev_waveData);	checkCudaError();
	//hipFree(dev_waveDataNormalised);
	
	printf("normalise wave done\n");
	
	/***********************Calling normalisingHammingKernel*********************************/
	//Hamming data
	double * dev_hamming;
	hipMalloc((void **)&dev_hamming, KEYS*KEYBYTES*SAMPLES*sizeof(double));	checkCudaError();
	hipMemcpy(dev_hamming, hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double), hipMemcpyHostToDevice);	checkCudaError();
	
	//Result matrices
	unsigned int * hammingNormalised = (unsigned int *)malloc(sizeof(unsigned int) * KEYS*KEYBYTES*SAMPLES);
	unsigned int * dev_hammingNormalised;
	hipMalloc((void **)&dev_hammingNormalised, KEYS*KEYBYTES*SAMPLES*sizeof(unsigned int));	checkCudaError();

	//Store secondNumState
	unsigned int * dev_secondNumState;
	hipMalloc((void **)&dev_secondNumState, sizeof(unsigned int)*KEYS*KEYBYTES);
	unsigned int * secondNumState = (unsigned int *)malloc(sizeof(unsigned int) * KEYS*KEYBYTES);
	
	dim3 numBlocksHamming(256, 1, 1);
	dim3 numThreadsHamming(16, 1, 1);
	
	normaliseHammingKernel<<<numBlocksHamming, numThreadsHamming>>>(dev_hamming, dev_hammingNormalised, dev_secondNumState);
	hipDeviceSynchronize();	checkCudaError();
	
	hipMemcpy(hammingNormalised, dev_hammingNormalised, KEYS*KEYBYTES*SAMPLES*sizeof(unsigned int), hipMemcpyDeviceToHost);	checkCudaError();
	hipMemcpy(secondNumState, dev_secondNumState, KEYS*KEYBYTES*sizeof(unsigned int), hipMemcpyDeviceToHost);	checkCudaError();
	
	hipFree(dev_hamming);	checkCudaError();
	printf("Hamming normalised, dev_hamming removed\n");
	
	/***************************************Calling findProbWave******************************************/
	int maxOfFirst = arrayMax(firstNumState, SAMPLEPOINTS);
	
	double *dev_firstStateProbs;
	hipMalloc((void **)&dev_firstStateProbs, SAMPLEPOINTS*maxOfFirst*sizeof(double));	checkCudaError();
	
	double *firstStateProbs = (double *)malloc(sizeof(double)*SAMPLEPOINTS*maxOfFirst);

	dim3 numBlocksProbsWave(16, 16, 1);
	dim3 threadsPerBlocksProbsWave(512, 1, 1);
	
	findProbsWave<<<numBlocksProbsWave, threadsPerBlocksProbsWave>>>(dev_waveDataNormalised, dev_firstStateProbs, dev_firstNumState, maxOfFirst);		checkCudaError(); 
	hipDeviceSynchronize(); checkCudaError();
	/***************************************Calling findProbsHamming******************************************/
	int maxOfSecond = arrayMax(secondNumState, KEYS*KEYBYTES);
	
	double *dev_secondStateProbs;
	
	hipMalloc((void **)&dev_secondStateProbs, KEYS*KEYBYTES*maxOfSecond*sizeof(double));	checkCudaError();
	double *secondStateProbs = (double *)malloc(sizeof(double)*KEYS*KEYBYTES*maxOfSecond);
	
	//we need 16 parallel operations
	dim3 numBlocksProbsHamming(256, 1, 1); 
	dim3 threadsPerBlocksProbsHamming(16,1,1);
	
	findProbsHamming<<<numBlocksProbsHamming, threadsPerBlocksProbsHamming>>>(dev_hammingNormalised, dev_secondStateProbs, dev_secondNumState, maxOfSecond);	checkCudaError();
	hipDeviceSynchronize();
	
	printf("Hamming probabilities calculated\n");
	hipMemcpy(secondStateProbs, dev_secondStateProbs, KEYS*KEYBYTES*maxOfSecond*sizeof(double), hipMemcpyDeviceToHost);
	/*************************************************************Calling joint Probs***************************************************/
	/*calculating joint sizes*/
	unsigned long *jointSizes = (unsigned long*)malloc((KEYS/32)*KEYBYTES*SAMPLEPOINTS*sizeof(unsigned long)); //for 8 keys @ a momment
	int i1,i2,i3;
	
	double *dev_MIvals;
	hipMalloc((void **)&dev_MIvals, KEYS*KEYBYTES*SAMPLEPOINTS*sizeof(double));   checkCudaError();
	hipMemset(dev_MIvals, 0, KEYS*KEYBYTES*SAMPLEPOINTS*sizeof(double));	checkCudaError();
	
	double *MIvals = (double *)malloc(KEYS*KEYBYTES*SAMPLEPOINTS*sizeof(double));
	
	unsigned long *dev_jointSizes;
	hipMalloc((void **)&dev_jointSizes, (KEYS/32)*KEYBYTES*SAMPLEPOINTS*sizeof(unsigned long)); 
	
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
	
	/*confirm the correctness of this*/
	dim3 block3d(16,2,4);
	dim3 grid3d(1,4,25000);
		
	int turn;
	for (turn=0;turn<8*32;turn+=8){
		
		unsigned long totalSize = 0l;
	
		for(i1=0;i1<(KEYS/32);i1++){
			for(i2=0;i2<KEYBYTES;i2++){
				for(i3=0;i3<SAMPLEPOINTS;i3++){
					totalSize += (long)firstNumState[i3]*(long)secondNumState[(turn+i1)*KEYBYTES + i2];
					if (totalSize>7466209652) printf("THis sucksssssss\n");
					jointSizes[i1*KEYBYTES*SAMPLEPOINTS + i2*SAMPLEPOINTS + i3] = totalSize;
				}
			}
		}
	
		unsigned long ttl = totalSize*8;
		//printf("This is the total size: %ld bytes\n",ttl);
		//free(jointSizes);
	
		hipMemcpy(dev_jointSizes, jointSizes, (KEYS/32)*KEYBYTES*SAMPLEPOINTS*sizeof(unsigned long), hipMemcpyHostToDevice);  checkCudaError();
	
		double *dev_jointProbs;
		hipMalloc((void **)&dev_jointProbs, totalSize*sizeof(double));   checkCudaError();
		
		//double *jointProbs = (double *)malloc(totalSize*sizeof(double));
	
		
	
		/**findJointProbs occupancy is maximum**/
	
		
	
		//hipMemset(dev_jointProbs, 0, (KEYS/4)*KEYBYTES*SAMPLES*sizeof(double));   checkCudaError(); //??
		hipMemset(dev_jointProbs, 0, totalSize*sizeof(double));   checkCudaError();
		
		printf("Now starting joint probs and MI calculation for turn = %d\n",turn);
		
		findJointProbs2<<<grid3d, block3d>>>(dev_MIvals, dev_waveDataNormalised, dev_hammingNormalised, 
											 dev_firstNumState, dev_secondNumState, dev_firstStateProbs, dev_secondStateProbs, dev_jointProbs, 
											 dev_jointSizes, maxOfFirst, maxOfSecond, turn); 		hipDeviceSynchronize();		checkCudaError();
	
		hipFree(dev_jointProbs);
	}	
	printf("MI cal done\n");
	
	//hipMemcpy(MIvals, dev_MIvals, KEYBYTES*SAMPLEPOINTS*sizeof(double), hipMemcpyDeviceToHost); checkCudaError();
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(stdout,"Time spent for operation : %.10f seconds\n",elapsedtime/(float)1000);
	
	
	free(firstStateProbs);
	free(secondStateProbs);
	
	
	firstStateProbs = NULL;
	secondStateProbs = NULL;
	
	free(plainTexts);
	free(hammingMat);
	free(waveData);
	free(waveDataNormalised);
	free(hammingNormalised);
	
	hipFree(dev_firstStateProbs);
	hipFree(dev_secondStateProbs);
	
	
	
	hipFree(dev_waveDataNormalised);
	printf("dev_waveDataNormalised done\n");

	hipFree(dev_hammingNormalised);
	printf("dev_hammingNormalised done\n");

	hipFree(dev_firstNumState);
	printf("dev_firstNumState done\n");

	hipFree(dev_secondNumState);
	printf("dev_secondNumState done\n");
	
	/***********Sorting***************/
	printf("sorting\n");
	double *dev_sortedMI;
	hipMalloc((void **)&dev_sortedMI, KEYBYTES*KEYS*sizeof(double)); 
	
	double *sortedMI = (double *)malloc(KEYBYTES*KEYS*sizeof(double));
	
	dim3 block3dS(16,16,1);
	dim3 grid3dS(1,16,1);
	
	simpleSort<<<grid3dS, block3dS>>>(dev_MIvals, SAMPLEPOINTS, dev_sortedMI); 	hipDeviceSynchronize(); 	checkCudaError();
	
	hipMemcpy(sortedMI, dev_sortedMI, KEYBYTES*KEYS*sizeof(double), hipMemcpyDeviceToHost); checkCudaError();
	
	FILE * fpP = freopen("results_MISorted.txt", "w", stdout);
	
	int pp1,pp2;
	
	for(pp1=0;pp1<KEYS;pp1++){
		for(pp2=0;pp2<KEYBYTES;pp2++){
			printf("%lf  ",sortedMI[pp1*KEYBYTES + pp2]);
		}
		printf("\n");
	}
	fclose(fpP);
	/****************************************************************************************************************************/
	
	hipFree(dev_MIvals);
	hipFree(dev_sortedMI);
	
	free(MIvals);
	free(sortedMI);
	
	printf("dev_MI_vals done\n");
	printf("*********************END OF PROGRAM*********************\n");
	return 0;
}
