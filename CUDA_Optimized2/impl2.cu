#include "hip/hip_runtime.h"
//
//	Version information:
//
//	Contains normalising functions for hammingMat and waveData
//
//	MP			Initial version
//	BW			Added makeAllHamming() method
//				Added printMatDouble() to arrayOperations.cu file
//				Changed hammingMat size
//				Changed argument order in hipMemcpyDeviceToHost
//				Added sampleNum argument to hamming() method
//				
//	Grid for hamming matrix calculation
//	------------------------------------------------------------------  
//	|	256	|	256	|	256	|	256	|	256	|	256	| .....	|	256	|		
//	------------------------------------------------------------------
//	<-------------------------- 16 blocks ---------------------------->
//
//	Hamming result vector
//	--------------------------------------------------------------------
//	|<-200->			|   				|					|		
//	|   0  |...|   15   |		|	|		|		|	|		|	
//	|<-----16*200----->	|					|					|	
//	--------------------------------------------------------------------
// <-------------------------- 16*200*256 ---------------------------->
//	index of an element = key*KEYBYTES*SAMPLES + keyByte*SAMPLES + sampleNumber

#include <stdlib.h>
#include <stdio.h>
#include <syslog.h>
#include "data.cuh"
#include "arrayOperations.cuh"
#include "probOps.cuh"
#include "helpers.cuh"

#define PLAINFILE "plain.txt"
//#define WAVEFILE "wave.txt"
#define WAVEFILE "/home/e11444/CO421/test2/wave.txt"

//number of sample points in one encryption
/*#define SAMPLEPOINTS 100000
#define SAMPLES 200

//defined for 128 bit AES
#define KEYBYTES 16
#define KEYS 256*/

//calculates hamming weight of a 8 bit number
__device__ byte hammingweight(byte H){

	// Count the number of set bits
	byte dist=0;
	while(H){
		dist++; 
		H &= H - 1;
	}
	return dist;
}

//find hamming weight for the selection function
/*__device__ byte hamming(unsigned int *sample, unsigned int i,unsigned int n,unsigned int key){ //n is byteno  i is the sample
    //byte inter = (byte)sbox[sample[i*KEYBYTES+n] ^ key];
    byte inter = (byte)sbox[sample[i] ^ key];
    //byte inter = 0x00;
    byte dist = hammingweight(inter);	  
	return dist;
}*/

__device__ byte hamming(unsigned int *sample, unsigned int index, unsigned int key, unsigned int sampleNum){ //n is byteno  i is the sample
	byte inter = sbox[sample[sampleNum*KEYBYTES+index] ^ key]; 
    //byte inter = 0xAB;
    byte dist = hammingweight(inter);	  
	return dist;
}

//__global__ void makeHamming(unsigned int *plaintxt, double *hammingRes, int n){//n is the size of samples 
	
	//unsigned int key = 0x01;
	//int index = blockIdx.x*KEYBYTES + threadIdx.x;
	
	//if (index<n){
		//byte temp = hamming(plaintxt,index, key);
		//hammingRes[index] = temp;
	//}
//} 

//----------------------------------------------------------------------------------------------------------
//	Method to calculate hamming weights of all keyBytes of all samples w.r.t. all key guesses
//----------------------------------------------------------------------------------------------------------
__global__ void makeAllHamming(unsigned int *plaintxt, double *hammingRes, int n){//n is the size of samples 
	
	int keyByte = threadIdx.y;
	int key = blockIdx.x*KEYBYTES + threadIdx.x;
	int i;

	if ((keyByte < KEYBYTES) && (key < KEYS)){
		
		for (i=0; i<SAMPLES; i++){
			double temp = (double)hamming(plaintxt,keyByte, key, i); 			
			hammingRes[key*KEYBYTES*SAMPLES + keyByte*SAMPLES + i] = temp;
		}
	}
} 

//--------------------------------------------------------------------------------------------------------
//	Find maximum of each MI set
//--------------------------------------------------------------------------------------------------------
__global__ void simpleSort(double *origMat, int cols, double *maxOfEach, int key){
	
	int index = threadIdx.x;
	int i;
	double max = 0.0;
	for (i=0;i<cols;i++){
		if (max<origMat[key*KEYBYTES*cols + index*cols + i])
			max = origMat[key*KEYBYTES*cols + index*cols + i];
	}
	maxOfEach[key*KEYBYTES + index] = max;
}

int arrayMax(unsigned int * data, int length){
	
	int max = data[0];
	for(int i=0; i<length; i++){
		if(data[i]>max)
			max = data[i];
	}
	return max;
}

int main(){

	//hipSetDevice(1);
	


	//plainText at host
	int width = 256*16;
	int i,j;
	unsigned int *dev_plainTexts;
	unsigned int *plainTexts = (unsigned int*)malloc(KEYBYTES*SAMPLES*sizeof(unsigned int));
	//checkMalloc(plainTexts);

	/*** Reading plaintext file ***/
	FILE *plainT = fopen(PLAINFILE,"r");
	char temp1[32];	

	for(i=0; i<SAMPLES ;i++){
		fscanf(plainT,"%s",temp1);
		for(j=0; j<KEYBYTES; j++){
			int post = i*KEYBYTES + j;
			sscanf(&temp1[2*j],"%02X",&plainTexts[post]);		
		}
	}
	
	/*** reading wave file ***/
	FILE *waveStuff = fopen(WAVEFILE,"r");
	
	double *waveData= (double *)malloc(sizeof(double) * SAMPLEPOINTS * SAMPLES);

	float dat;

	
	for(i=0; i<SAMPLES ;i++){
		for(j=0; j<SAMPLEPOINTS; j++){
			fscanf(waveStuff,"%f",&dat); //?? 
			int pos = j*SAMPLES + i;
			//waveData[j][i]=(double)dat*1000.0;
			waveData[pos]=(double)dat*1000.0;
		}
	}
	
	fclose(waveStuff);
	
	//openlog("CUDAmia", NULL, 0);
	printf("WIDTH: %d\n", WIDTH);
	/****************************Calling makeAllHamming kernel*************************/
	
	//hammingMat at host
	//double *hammingMat = (double *)malloc(KEYBYTES*SAMPLES*sizeof(double));
	double *hammingMat = (double *)malloc(KEYS*KEYBYTES*SAMPLES*sizeof(double));
	double *dev_hammingMat;
	hipMalloc((void **)&dev_plainTexts, KEYBYTES*SAMPLES*sizeof(unsigned int));
	hipMemcpy(dev_plainTexts, plainTexts, KEYBYTES*SAMPLES*sizeof(unsigned int), hipMemcpyHostToDevice);
	//hipMalloc((void **)&dev_hammingMat, KEYBYTES*SAMPLES*sizeof(double));
	hipMalloc((void **)&dev_hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double));
	
	dim3 numBlocks(16,1); //Blocks in the grid
	dim3 numThreads(16,16); // Threads per block
	
	makeAllHamming<<<numBlocks,numThreads>>>(dev_plainTexts, dev_hammingMat, width);
	hipDeviceSynchronize();
	
	hipMemcpy(hammingMat, dev_hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(dev_hammingMat);
	hipFree(dev_plainTexts);
	printf("make hamming done\n");
	
	/****************** Calling normaliseWaveDataKernel on waveData ********************/

	//Store wave data in device
	double * dev_waveData;
	hipMalloc((void **)&dev_waveData, SAMPLEPOINTS * SAMPLES*sizeof(double));
	hipMemcpy(dev_waveData, waveData, SAMPLEPOINTS * SAMPLES*sizeof(double), hipMemcpyHostToDevice);
	
	//Store normalised wave data
	unsigned int * waveDataNormalised = (unsigned int *)malloc(sizeof(unsigned int) * SAMPLEPOINTS * SAMPLES);
	unsigned int * dev_waveDataNormalised;
	hipMalloc((void **)&dev_waveDataNormalised, SAMPLEPOINTS * SAMPLES*sizeof(unsigned int));
	
	//Store firstNumState
	unsigned int * dev_firstNumState;
	hipMalloc((void **)&dev_firstNumState, sizeof(unsigned int)*SAMPLEPOINTS);
	unsigned int * firstNumState = (unsigned int *)malloc(sizeof(unsigned int) * SAMPLEPOINTS);
	
	dim3 numBlocksNorm(100, 1, 1);
	dim3 numThreadsNorm(1000, 1, 1);
	
	normaliseWaveDataKernel<<<numBlocksNorm, numThreadsNorm>>>(dev_waveData, dev_waveDataNormalised, dev_firstNumState);
	hipDeviceSynchronize();
	
	hipMemcpy(waveDataNormalised, dev_waveDataNormalised, SAMPLEPOINTS*SAMPLES*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(firstNumState, dev_firstNumState, SAMPLEPOINTS*sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	hipFree(dev_waveData);
	//hipFree(dev_waveDataNormalised);
	
	printf("normalise wave done\n");
	
	/***********************Calling normalisingHammingKernel*********************************/
	//Hamming data
	double * dev_hamming;
	hipMalloc((void **)&dev_hamming, KEYS*KEYBYTES*SAMPLES*sizeof(double));
	hipMemcpy(dev_hamming, hammingMat, KEYS*KEYBYTES*SAMPLES*sizeof(double), hipMemcpyHostToDevice);
	
	//Result matrices
	unsigned int * hammingNormalised = (unsigned int *)malloc(sizeof(unsigned int) * KEYS*KEYBYTES*SAMPLES);
	unsigned int * dev_hammingNormalised;
	hipMalloc((void **)&dev_hammingNormalised, KEYS*KEYBYTES*SAMPLES*sizeof(unsigned int));

	//Store secondNumState
	unsigned int * dev_secondNumState;
	hipMalloc((void **)&dev_secondNumState, sizeof(unsigned int)*KEYS*KEYBYTES);
	unsigned int * secondNumState = (unsigned int *)malloc(sizeof(unsigned int) * KEYS*KEYBYTES);
	
	dim3 numBlocksHamming(256, 1, 1);
	dim3 numThreadsHamming(16, 1, 1);
	
	normaliseHammingKernel<<<numBlocksHamming, numThreadsHamming>>>(dev_hamming, dev_hammingNormalised, dev_secondNumState);
	hipDeviceSynchronize();
	
	hipMemcpy(hammingNormalised, dev_hammingNormalised, KEYS*KEYBYTES*SAMPLES*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(secondNumState, dev_secondNumState, KEYS*KEYBYTES*sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	hipFree(dev_hamming);
	printf("Hamming normalised, dev_hamming removed\n");
	
	/***************************************Calling findProbWave******************************************/
	int maxOfFirst = arrayMax(firstNumState, SAMPLEPOINTS);
	
	double *dev_firstStateProbs;
	hipMalloc((void **)&dev_firstStateProbs, SAMPLEPOINTS*maxOfFirst*sizeof(double));	checkCudaError();
	
	double *firstStateProbs = (double *)malloc(sizeof(double)*SAMPLEPOINTS*maxOfFirst);

	dim3 numBlocksProbsWave(16, 16, 1);
	dim3 threadsPerBlocksProbsWave(512, 1, 1);
	
	findProbsWave<<<numBlocksProbsWave, threadsPerBlocksProbsWave>>>(dev_waveDataNormalised, dev_firstStateProbs, dev_firstNumState, maxOfFirst);		checkCudaError(); 
	hipDeviceSynchronize();
	/***************************************Calling findProbsHamming******************************************/
	int maxOfSecond = arrayMax(secondNumState, KEYS*KEYBYTES);
	
	double *dev_secondStateProbs;
	
	hipMalloc((void **)&dev_secondStateProbs, KEYS*KEYBYTES*maxOfSecond*sizeof(double));	checkCudaError();
	double *secondStateProbs = (double *)malloc(sizeof(double)*KEYS*KEYBYTES*maxOfSecond);
	
	//we need 16 parallel operations
	dim3 numBlocksProbsHamming(256, 1, 1); 
	dim3 threadsPerBlocksProbsHamming(16,1,1);
	
	findProbsHamming<<<numBlocksProbsHamming, threadsPerBlocksProbsHamming>>>(dev_hammingNormalised, dev_secondStateProbs, dev_secondNumState, maxOfSecond);	checkCudaError();
	hipDeviceSynchronize();
	
	printf("Hamming probabilities calculated\n");
	hipMemcpy(secondStateProbs, dev_secondStateProbs, KEYS*KEYBYTES*maxOfSecond*sizeof(double), hipMemcpyDeviceToHost);
	
	//FILE * fpP = freopen("results_prob_hamming.txt", "w", stdout);
	//printMatDouble(secondStateProbs,100,maxOfSecond*KEYBYTES);
	//fclose(fpP);
	/*************************************************************Calling joint Probs***************************************************/
	int maxOfJoint = maxOfFirst*maxOfSecond;
	printf("Max of first: %d\n",maxOfFirst);
	printf("Max of second: %d\n",maxOfSecond);
	printf("Max of joint: %d\n",maxOfJoint);

	double *dev_jointProbs;
	
	int threads = SAMPLEPOINTS/REPEAT;
	unsigned int index = 0;
	
	//hipMalloc((void **)&dev_jointProbs, KEYBYTES*SAMPLEPOINTS*maxOfJoint*sizeof(double));
	hipMalloc((void **)&dev_jointProbs, KEYBYTES*threads*maxOfJoint*sizeof(double)); 
	double *jointProbs = (double *)malloc(KEYBYTES*threads*maxOfJoint*sizeof(double));
	
	//double *dev_MIvals;
	//hipMalloc((void **)&dev_MIvals, KEYBYTES*SAMPLEPOINTS*sizeof(double)); 
	//double *MIvals = (double *)malloc(KEYBYTES*SAMPLEPOINTS*sizeof(double));
	
	double *dev_MIvals;
	hipMalloc((void **)&dev_MIvals, KEYS*KEYBYTES*SAMPLEPOINTS*sizeof(double)); 
	hipMemset(dev_MIvals, 0, KEYS*KEYBYTES*SAMPLEPOINTS*sizeof(double));
	double *MIvals = (double *)malloc(KEYS*KEYBYTES*SAMPLEPOINTS*sizeof(double));
	
	//dim3 numBlocksJointProb(128,16,1);
	//dim3 threadsPerBlockJointProb(1024,1,1);
	dim3 numBlocksJointProb(32,16,1);
	dim3 threadsPerBlockJointProb(256,1,1); //OPTIMUM: 256 threads per block. Occupancy reduce for lesser value.
	
	/**findJointProbs occupancy is maximum**/
	
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
	
	for(i=0; i<NUMOFKEYS; i++){
		int startPosition = 0;
		for(j=0; j<REPEAT; j++){
			//printf("This is the Start position: %d\n",startPosition);
			
			hipMemset(dev_jointProbs, 0, KEYBYTES*threads*maxOfJoint*sizeof(double));
			
			
			findJointProbs<<<numBlocksJointProb, threadsPerBlockJointProb>>>(dev_MIvals, dev_waveDataNormalised, dev_hammingNormalised, dev_firstNumState, 
						dev_secondNumState, dev_firstStateProbs, dev_secondStateProbs, dev_jointProbs, 
						maxOfFirst, maxOfSecond, i, startPosition); 	
			hipDeviceSynchronize();
			checkCudaError();
			//if(i==REPEAT-1)	
			//	hipMemcpy(jointProbs, dev_jointProbs, (KEYBYTES*threads*maxOfJoint)*sizeof(double), hipMemcpyDeviceToHost); checkCudaError();
			
			startPosition = startPosition + threads;
			//index = index + (KEYBYTES*threads*maxOfJoint);
			//printf("index: %ul\n", index);
		}
	}
	
	printf("MI cal done\n");
	hipMemcpy(MIvals, dev_MIvals, KEYBYTES*SAMPLEPOINTS*sizeof(double), hipMemcpyDeviceToHost); checkCudaError();
	
	/*int oo,aa;
	for(oo=0;oo<2;oo++){
		for(aa=0;aa<20;aa++){
			printf("MI[%d] = %lf\n",(oo*SAMPLEPOINTS + aa),MIvals[oo*SAMPLEPOINTS + aa]);
		}
		printf("==================================================\n");
	}*/
	
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(stdout,"Time spent for operation : %.10f seconds\n",elapsedtime/(float)1000);
	
	/***********Sorting***************/
	double *dev_sortedMI;
	hipMalloc((void **)&dev_sortedMI, KEYBYTES*KEYS*sizeof(double)); 
	hipMemset(dev_sortedMI, 0, KEYBYTES*KEYS*sizeof(double));
	double *sortedMI = (double *)malloc(KEYBYTES*KEYS*sizeof(double));
	
	dim3 numBlockSort(1,1,1);
	dim3 threadsPerBlockSort(16,1,1);
	
	for (i=0;i<NUMOFKEYS;i++){
		simpleSort<<<numBlockSort, threadsPerBlockSort>>>(dev_MIvals, 100000, dev_sortedMI, i); checkCudaError();
	}
	hipMemcpy(sortedMI, dev_sortedMI, KEYBYTES*KEYS*sizeof(double), hipMemcpyDeviceToHost); checkCudaError();
	
	
	
	FILE * fpP = freopen("results_MISorted.txt", "w", stdout);
	int pp1,pp2;
	for(pp1=0;pp1<NUMOFKEYS;pp1++){
		for(pp2=0;pp2<KEYBYTES;pp2++){
			printf("%lf  ",sortedMI[pp1*KEYBYTES + pp2]);
		}
		printf("\n");
	}
	fclose(fpP);
	/****************************************************************************************************************************/
	free(firstStateProbs);
	free(secondStateProbs);
	free(jointProbs);
	
	firstStateProbs = NULL;
	secondStateProbs = NULL;
	jointProbs = NULL;
	
	free(plainTexts);
	free(hammingMat);
	free(waveData);
	free(waveDataNormalised);
	free(hammingNormalised);
	free(MIvals);
	free(sortedMI);
	
	hipFree(dev_firstStateProbs);
	hipFree(dev_secondStateProbs);
	
	hipFree(dev_jointProbs);
	
	hipFree(dev_waveDataNormalised);
	printf("dev_waveDataNormalised done\n");

	hipFree(dev_hammingNormalised);
	printf("dev_hammingNormalised done\n");

	hipFree(dev_firstNumState);
	printf("dev_firstNumState done\n");

	hipFree(dev_secondNumState);
	printf("dev_secondNumState done\n");
	
	hipFree(dev_MIvals);
	printf("dev_secondNumState done\n");
	printf("*********************END OF PROGRAM*********************\n");
	
	hipFree(dev_sortedMI);
	
	return 0;
}
